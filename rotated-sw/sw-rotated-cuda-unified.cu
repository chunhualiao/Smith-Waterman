#include "hip/hip_runtime.h"

/// \brief
/// This implements the Smith-Waterman algorithm based on a 45 degree 
/// rotated dynamic programming matrix. The benefit of the
/// rotation is:
///   - each diagonal can be represented by contiguous memory
///     --> reduces page faults
///     --> reduces resident memory use in the core algorithm
///     --> enables the algorithm to run more efficiently on large
///         inputs.
///   
/// The disadvantages of the rotation is:
///   - if the original output matrix needs to be maintained, copying
///     back may lead to costly page faults, outweighing the benefits of 
///     the rotation.
///     see @todo_1 for details.
///
/// \email pirkelbauer2@llnl.gov

/*
 * Compilation: nvcc -std=c++11 -O3 -DNDEBUG=1 sw-rotated-cuda-unified.cu -o smithW-cuda
 *              nvcc -std=c++11 -O0 -G -g sw-rotated-cuda-unified.cu -o dbg-smithW-cuda
 */

#include <vector>
#include <limits>
#include <cassert>
#include <algorithm>
#include <utility>
#include <iostream>
#include <chrono>

#include "parameters.h"

static const bool DEBUG_MODE = false;

/*--------------------------------------------------------------------
 * Text Tweaks
 */
#define RESET   "\033[0m"
#define BOLDRED "\033[1m\033[31m"      /* Bold Red */
/* End of text tweaks */

/// defines type for indices into arrays and matrices
///    (needs to be a signed type)
typedef long long int index_t;

/// defines data type for scoring
typedef int           score_t;

/// defines data type for linking paths
enum link_t { UNDEF = -1, NOLINK = 0, UP = 1, LEFT = 2, DIAGONAL = 3 };

// global constants
static const score_t PATH            = -1;
static const score_t NONE            =  0; // -4
static const score_t MATCH_SCORE     =  3; //  5 in omp_smithW_orig
static const score_t MISSMATCH_SCORE = -3; // -3
static const score_t GAP_SCORE       = -2; // -4

typedef std::vector<char> char_seq;

template <class T>
static inline
void rotate3(T& a, T& b, T& c)
{
  T tmp = a;

  a = c; c = b; b = tmp;
}


static inline
__device__
int matchMissmatchScore(const char* a, const char* b, size_t ai, size_t bi)
{
  return a[ai] == b[bi] ? MATCH_SCORE : MISSMATCH_SCORE;
}  /* End of matchMissmatchScore */


static
__global__
void similarityScore_kernel( index_t iterspace_lb,
                             index_t iterspace_ub,
                             index_t i,
                             score_t* M_0,
                             link_t* P_0,
                             const score_t* M_1,
                             const score_t* M_2,
                             const char* a,
                             const char* b,
                             const score_t** maxpos
                           )
{
  const index_t loop_j = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (loop_j >= iterspace_ub - iterspace_lb) return;
   
  const index_t ai   = (iterspace_ub - loop_j) - 1;
  const index_t bi   = i - ai;  
  const index_t j    = iterspace_lb + loop_j;    
  
  assert(!DEBUG_MODE || (M_1[j] >= 0 && M_1[j-1] >= 0 && M_2[j-1] >= 0));
  
  const index_t up   = M_1[j]   + GAP_SCORE;
  const index_t lft  = M_1[j-1] + GAP_SCORE;
  const index_t diag = M_2[j-1] + matchMissmatchScore(a, b, ai-1, bi-1);
  
  score_t       max  = NONE;
  link_t        pred = NOLINK;

  if (up > max)
  {
    max  = up;
    pred = UP;
  }

  if (lft > max)
  {
    max  = lft;
    pred = LEFT;
  }

  if (diag > max)
  {
    max  = diag;
    pred = DIAGONAL;
  }

  assert(!DEBUG_MODE || (M_0[j] < 0));
  M_0[j] = max;
  P_0[j] = pred;
  
  // Updates maximum score to be used as seed on backtrack
  {   
    // \note \pp
    //   locks seem to be a NOGO in CUDA warps,
    //   thus the update to set the maximum is made nonblocking.
    const score_t* assumed = nullptr;
    const score_t* current = *maxpos;
  
    while ((current != assumed) && max > *current)
    {
      // \note consider atomicCAS_system for multi GPU systems
      assumed = current;      
      current = (const score_t*) atomicCAS( (unsigned long long int*) maxpos, 
                                            (unsigned long long int) assumed, 
                                            (unsigned long long int) (M_0 + j)
                                          );
    }
  }
}

/*--------------------------------------------------------------------
 * Function:    calcElement
 * Purpose:     Calculate the first element of a given diagonal
 */
index_t diagonalBasePoint(index_t i, index_t w)
{
  // base point is on the first row
  if (i-1 <= w) return i-1;

  // base point is the last element on the (i-w)+2 th row
  return (w+1)*(i-w)-1;
}


static inline
void check_cuda_success(hipError_t err)
{
  if (err == hipSuccess) return;

  std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
  exit(0);
}

/// malloc replacement
template<class T>
static
T* unified_alloc(size_t numelems)
{
  void*       ptr /* = NULL*/;
  hipError_t err = hipMallocManaged(&ptr, numelems * sizeof(T), hipMemAttachGlobal);

  check_cuda_success(err);
  return reinterpret_cast<T*>(ptr);
}

static
void unified_free(void* ptr)
{
  hipError_t err = hipFree(ptr);

  check_cuda_success(err);
}


/// \brief computes smith-waterman
/// \param a input sequence of length w
/// \param b input sequence of length h
/// \param w length of input sequence a
/// \param h length of input sequence b
/// \param H output matrix (size == (w+1) * (h+1)) representing all scores
/// \param P output matrix (size == (w+1) * (h+1)) to link longest sequences
/// \param maxscore output score of longest matching sequence in H and P
/// \param maxloc output position of longest matching sequence in H and P
/// \note output data does not need to be initialized
void smithWaterman( const char* a,
                    const char* b,
                    index_t w,
                    index_t h,
                    score_t* H,
                    link_t* P,
                    score_t** maxloc
                  )
{
  // Size is important for pointer CAS in CUDA Kernel
  static_assert( sizeof(maxloc) == sizeof(unsigned long long int), 
                 "pointer/int size mismatch (req. for CUDA atomicCAS)!"
               );

  const index_t  MAXITER  = 2 + w + h - 1;

  // wavefront arrays for three iterations
  score_t* const        wavefronts = unified_alloc<score_t>(3*MAXITER);
  link_t*               pred_0     = unified_alloc<link_t>(MAXITER);
  link_t*               pred_1     = unified_alloc<link_t>(MAXITER);
  score_t* const        maxscr     = unified_alloc<score_t>(1);
  const score_t** const maxpos     = unified_alloc<const score_t*>(1);
  
  // wavefront representation _time
  score_t*       M_2 = wavefronts;
  score_t*       M_1 = wavefronts + MAXITER;

  // wavefront output
  score_t*       M_0 = wavefronts + 2*MAXITER;

  // initialize t == 0
  M_1[0] = NONE;

  // set maxloc to origin, and origin to 0
  *maxloc  = H;
  **maxloc = 0;
  *maxscr  = 0;

  // smith waterman
  for (index_t i = 1; i <= MAXITER; ++i)
  {
    const index_t  lb       = (i<=h) ? (M_0[0] = NONE, 1) : i - h;
    const index_t  ub       = (i<=w) ? (M_0[i] = NONE, i) : w + 1;
    
    *maxpos = maxscr;
    
    assert((ub - lb >= 0) && (ub - lb <= h));
    
    const index_t THREADS_PER_BLOCK = 1024;
    const index_t ITER_SPACE = (ub-lb+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
    
    similarityScore_kernel
      <<<ITER_SPACE, THREADS_PER_BLOCK>>>
      (lb, ub, i, M_0, pred_0, M_1, M_2, a, b, maxpos);      

    // rotate wavefront vectors
    rotate3(M_0, M_1, M_2);

    // swap output vectors
    std::swap(pred_0, pred_1);

    hipDeviceSynchronize();

    // for debugging purposes clear all M_0
    if (DEBUG_MODE)
      std::fill(M_0, M_0+MAXITER, std::numeric_limits<score_t>::min());

    // \note
    // The problem with the CUDA algorithm is that getting the
    // data off of the GPU produces an enormous amount of page faults.
    // e.g., on Lassen with a 100000x10000 data set copying the data
    //       out from the GPU results in a 20x performance hit.
    //
    // \todo @todo_1
    // Can we hide the data transfer behind the computation of the
    // next iteration?
    // - in principle, this should be possible. The next round of
    //   computation only changes M_0, and pred_0, thus we could
    //   overlap the computation with data transfer back to the CPU.
    // - in a first attempt, CUDA streams were attempted. In this
    //   versions there existed a computestream and a transferstream.
    //   the transferstream synchronized one time step after the
    //   computation. However, the result was a minor slow down to
    //   the version in this file.
    //   see file sw-rotated-cuda-global-stream.cu
    //
    // - TRY to copy the data back into the H,P arrays using OpenMP
    //   ...
    // - Alternatively, the interface to SmithWaterman could be changed.
    //   The H,P representations are artifacts from the standard
    //   algorithm. By moving towards a 45 degree rotated base version
    //   copying back could be entirely avoided at the expense that
    //   the data representation becomes twice is big (though this
    //   could possibly be remedied by using a clever data layout).
    if (0)
    {
      // results from this iteration are in M_1
      // -> strided copies back to H and P
      index_t       ofs  = diagonalBasePoint(i, w) + w + 1;      
      hipError_t   errH = hipMemcpy2D( H+ofs, 
                                         w*sizeof(*H), 
                                         M_1 + lb, 
                                         sizeof(*M_1), 
                                         sizeof(*M_1), 
                                         ub-lb, 
                                         hipMemcpyDefault
                                       );      
      check_cuda_success(errH);
      
      hipError_t errP  = hipMemcpy2D( P+ofs, 
                                        w*sizeof(*P), 
                                        pred_1 + lb, 
                                        sizeof(*pred_1), 
                                        sizeof(*pred_1), 
                                        ub-lb, 
                                        hipMemcpyDefault
                                      );      
      check_cuda_success(errP);
    }
    

    {
      const score_t* maxx = *maxpos;
            
      // update maxscore, if maxpos points to an improved location
      if (maxx != maxscr)
      {
        index_t       j   = maxx - M_1;
        index_t       ofs = diagonalBasePoint(i, w) + w + 1;

        ofs += (j - lb) * w;

        *maxloc = H + ofs;
        *maxscr = *maxx;
      }
    }
  }


  unified_free(wavefronts);
  unified_free(pred_0);
  unified_free(pred_1);
  unified_free(maxscr);
  unified_free(maxpos);
}


/*--------------------------------------------------------------------
 * Function:    backtrack
 * Purpose:     Modify matrix to print, path change from value to PATH
 */
void backtrack(link_t* P, index_t maxPos, index_t m) {
    //hold maxPos value
    index_t predPos = 0;

    //backtrack from maxPos to startPos = 0
    do {
        switch (P[maxPos])
        {
          case DIAGONAL:
            predPos = maxPos - m - 1;
            break;

          case UP:
            predPos = maxPos - m;
            break;

          case LEFT:
            predPos = maxPos - 1;
            break;

          default:
            assert(false);
        }

        P[maxPos] = static_cast<link_t>(P[maxPos] * PATH);
        maxPos = predPos;
    } while (P[maxPos] != NONE);
}  /* End of backtrack */

/*--------------------------------------------------------------------
 * Function:    printMatrix
 * Purpose:     Print Matrix
 */
void printMatrix(score_t* matrix, const char* a, const char* b, index_t m, index_t n) {
    printf("-\t-\t");
    for (index_t j = 0; j < m; j++) {
      printf("%c\t", a[j]);
    }
    printf("\n-\t");
    for (index_t i = 0; i < n+1; i++) { // Lines
        for (index_t j = 0; j < m+1; j++) {
          if (j==0 && i>0) printf("%c\t", b[i-1]);
            printf("%d\t", std::max(0, matrix[(m+1) * i + j]));
        }
        printf("\n");
    }

}  /* End of printMatrix */

/*--------------------------------------------------------------------
 * Function:    printPredecessorMatrix
 * Purpose:     Print predecessor matrix
 */
void printPredecessorMatrix(link_t* matrix, const char* a, const char* b, index_t m, index_t n) {
    printf("    ");
    for (index_t j = 0; j < m; j++) {
      printf("%c ", a[j]);
    }
    printf("\n  ");
    for (index_t i = 0; i < n+1; i++) { //Lines
        for (index_t j = 0; j < m+1; j++) {
          if (j==0 && i>0) printf("%c ", b[i-1]);
            index_t index = m * i + j;
            if (matrix[index] < 0) {
                printf(BOLDRED);
                if (matrix[index] == -UP)
                    printf("↑ ");
                else if (matrix[index] == -LEFT)
                    printf("← ");
                else if (matrix[index] == -DIAGONAL)
                    printf("↖ ");
                else
                    printf("- ");
                printf(RESET);
            } else {
                if (matrix[index] == UP)
                    printf("↑ ");
                else if (matrix[index] == LEFT)
                    printf("← ");
                else if (matrix[index] == DIAGONAL)
                    printf("↖ ");
                else
                    printf("- ");
            }
        }
        printf("\n");
    }

}  /* End of printPredecessorMatrix */

/*--------------------------------------------------------------------
 * Function:    generate
 * Purpose:     Generate arrays a and b
 */
void generate(char* a, char* b, index_t m, index_t n) {
    //Random seed
    srand(time(NULL));

    //Generates the values of a
    long long int i;
    for (i = 0; i < m; i++) {
        int aux = rand() % 4;
        if (aux == 0)
            a[i] = 'A';
        else if (aux == 2)
            a[i] = 'C';
        else if (aux == 3)
            a[i] = 'G';
        else
            a[i] = 'T';
    }

    //Generates the values of b
    for (i = 0; i < n; i++) {
        int aux = rand() % 4;
        if (aux == 0)
            b[i] = 'A';
        else if (aux == 2)
            b[i] = 'C';
        else if (aux == 3)
            b[i] = 'G';
        else
            b[i] = 'T';
    }
} /* End of generate */


/*--------------------------------------------------------------------
 * Function:    main
 */
int main(int argc, char* argv[])
{
  typedef std::chrono::time_point<std::chrono::system_clock> time_point;

  bool     useBuiltInData = true;
  index_t  m = 8;
  index_t  n = 9;

  if (argc==3)
  {
    m = strtoll(argv[1], NULL, 10);
    n = strtoll(argv[2], NULL, 10);
    useBuiltInData = false;
  }

  if (useBuiltInData)
    printf ("Using built-in data for testing ..\n");

  printf("Problem size: Matrix[%lld][%lld], FACTOR=%d CUTOFF=%d\n", n, m, FACTOR, CUTOFF);

  // Allocates a and b
  // \pp \note m (instead of m+1), b/c end marker is not needed
  //~ char* a = (char*)malloc(m * sizeof(char));
  //~ char* b = (char*)malloc(n * sizeof(char));
  char* a = unified_alloc<char>(m);
  char* b = unified_alloc<char>(n);

  std::cerr << "a,b allocated: " << m << "/" << n << std::endl;

  //~ // Because now we have zeros
  // \pp m and n are the lengths of input strings ..
  //~ m++;
  //~ n++;

  if (useBuiltInData)
  {
    //Uncomment this to test the sequence available at
    //http://vlab.amrita.edu/?sub=3&brch=274&sim=1433&cnt=1
    // assert(m=11 && n=7);
    // a[0] =   'C';
    // a[1] =   'G';
    // a[2] =   'T';
    // a[3] =   'G';
    // a[4] =   'A';
    // a[5] =   'A';
    // a[6] =   'T';
    // a[7] =   'T';
    // a[8] =   'C';
    // a[9] =   'A';
    // a[10] =  'T';

    // b[0] =   'G';
    // b[1] =   'A';
    // b[2] =   'C';
    // b[3] =   'T';
    // b[4] =   'T';
    // b[5] =   'A';
    // b[6] =   'C';
    // https://en.wikipedia.org/wiki/Smith%E2%80%93Waterman_algorithm#Example
    // Using the wiki example to verify the results
    assert(m==8 && n==9);

    b[0] =   'G';
    b[1] =   'G';
    b[2] =   'T';
    b[3] =   'T';
    b[4] =   'G';
    b[5] =   'A';
    b[6] =   'C';
    b[7] =   'T';
    b[8] =   'A';

    a[0] =   'T';
    a[1] =   'G';
    a[2] =   'T';
    a[3] =   'T';
    a[4] =   'A';
    a[5] =   'C';
    a[6] =   'G';
    a[7] =   'G';
  }
  else
  {
    // Gen random arrays a and b
    generate(a, b, m, n);
  }

  time_point     starttime = std::chrono::system_clock::now();

  // Allocates similarity matrix H
  score_t* H = (score_t*) calloc((m+1) * (n+1), sizeof(score_t));

  // Allocates predecessor matrix P
  link_t*  P = (link_t*)  calloc((m+1) * (n+1), sizeof(link_t));
  score_t* maxloc = nullptr;

  smithWaterman(a, b, m, n, H, P, &maxloc);

  time_point     endtime = std::chrono::system_clock::now();

  if (DEBUG_MODE)
  {
    printf("\nSimilarity Matrix:\n");
    printMatrix(H, a, b, m, n);

    printf("\nPredecessor Matrix:\n");
    printPredecessorMatrix(P, a, b, m, n);
  }

  if (useBuiltInData)
  {
    printf ("Verifying results using the builtinIn data: %s\n", (H[(n+1)*(m+1)-1]==7)?"true":"false");
    assert (H[(n+1)*(m+1)-1]==7);
  }

  backtrack(P, maxloc - H, m+1);

  int elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(endtime-starttime).count();

  printf("\nElapsed time: %d ms\n\n", elapsed);

  // Frees similarity matrixes
  free(H);
  free(P);

  //Frees input arrays
  //~ free(a);
  //~ free(b);  
  unified_free(a);
  unified_free(b);

  return 0;
}  /* End of main */
