/*********************************************************************************
 * Smith–Waterman algorithm
 * Purpose:     Local alignment of nucleotide or protein sequences
 * Authors:     Daniel Holanda, Hanoch Griner, Taynara Pinheiro
 * Compilation: nvcc -std=c++11 -O3 -DNDEBUG=1 cuda_global_smithW.cu -o cuda_gm_smithW
 *              nvcc -std=c++11 -O0 -DDEBUG -g -G cuda_shared_smithW.cu -o dbg_cuda_smithW
 * Execution:   ./cuda_smithW <number_of_col> <number_of_rows>
 *********************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
//~ #include <time.h>
//~ #include <omp.h>

#include <cassert>
#include <chrono>
#include <iostream>

#ifndef NDEBUG
static constexpr bool DEBUG_MODE = true;
#else
static constexpr bool DEBUG_MODE = false;
#endif /* NDEBUG */


/*--------------------------------------------------------------------
 * Text Tweaks
 */
#define RESET   "\033[0m"
#define BOLDRED "\033[1m\033[31m"      /* Bold Red */
/* End of text tweaks */

/*--------------------------------------------------------------------
 * Constants
 */
#define PATH -1
#define NONE 0
/* End of constants */

/*--------------------------------------------------------------------
* Helpers
*/
#define min(x, y) (((x) < (y)) ? (x) : (y))
#define max(a,b) ((a) > (b) ? a : b)

// #define DEBUG
/* End of Helpers */

// my types
// \note changed type to unsigned to make it collaborate with CUDA atomicCAS
// \todo maybe rename it to index_t and change all long longs to index_t
typedef unsigned long long maxpos_t;

/// defines type for indices into arrays and matrices
///    (needs to be a signed type)
typedef long long int index_t;

/// defines data type for scoring
typedef int           score_t;

/// defines data type for linking paths
enum link_t { UNDEF = -1, NOLINK = 0, UP = 1, LEFT = 2, DIAGONAL = 3 };



/*--------------------------------------------------------------------
 * Functions Prototypes
 */
int backtrack(link_t* P, maxpos_t maxPos);
void printMatrix(score_t* matrix);
void printPredecessorMatrix(link_t* matrix);
void generate(void);
long long int nElement(long long int i);

// \pp modified to pass i (a induction variable) by value
void calcFirstDiagElement(long long int i, long long int *si, long long int *sj);

/* End of prototypes */

/*--------------------------------------------------------------------
 * Global Variables
 */
// Defines size of strings to be compared
index_t m = 8; // Columns - Size of string a
index_t n = 9; // Rows    - Size of string b

// Defines scores
static const int       MATCH_SCORE     =  3; //  5 in omp_smithW_orig
static const int       MISSMATCH_SCORE = -3; // -3
static const int       GAP_SCORE       = -2; // -4

// GPU THREADS PER BLOCK
static const long long THREADS_PER_BLOCK = 1024;

// Strings over the Alphabet Sigma
char *a, *b;

/* End of global variables */

static inline
void check_cuda_success(hipError_t err)
{
  if (err == hipSuccess) return;

  std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
  exit(0);
}


template <class T>
T* shared_alloc_only(size_t numelems)
{
  void*       ptr /* = NULL*/;
  hipError_t err = hipMalloc(&ptr, numelems * sizeof(T));
  check_cuda_success(err);

  return reinterpret_cast<T*>(ptr);
}

template <class T>
T* shared_alloc(T*, size_t numelems)
{
  void*       ptr /* = NULL*/;
  hipError_t err = hipMalloc(&ptr, numelems * sizeof(T));
  check_cuda_success(err);

  return reinterpret_cast<T*>(ptr);
}

template <class T>
T* shared_alloc_transfer(T* src, size_t numelems = 1)
{
  T*          dst = shared_alloc_only<T>(numelems);
  hipError_t err = hipMemcpyAsync(dst, src, numelems * sizeof(T), hipMemcpyHostToDevice);
  check_cuda_success(err);

  return dst;
}

template <class T>
T* shared_alloc_zero(size_t numelems = 1)
{
  T* ptr = shared_alloc_only<T>(numelems);

  hipError_t err = hipMemsetAsync(ptr, 0, numelems*sizeof(T), 0);
  check_cuda_success(err);

  return ptr;
}


void shared_free(void* p)
{
  hipError_t err = hipFree(p);
  check_cuda_success(err);
}

template <class T>
void shared_to_host_free(T* dst, T* src, size_t numelems = 1)
{
  hipError_t err = hipMemcpyAsync(dst, src, numelems * sizeof(T), hipMemcpyDeviceToHost, 0);
  check_cuda_success(err);

  shared_free(src);
}


/*--------------------------------------------------------------------
 * Function:    matchMissmatchScore
 * Purpose:     Similarity function on the alphabet for match/missmatch
 */
__device__
score_t 
matchMissmatchScore_cuda(index_t i, index_t j, const char* seqa, const char* seqb)
{
    if (seqa[j - 1] == seqb[i - 1])
        return MATCH_SCORE;

    return MISSMATCH_SCORE;
}  /* End of matchMissmatchScore_cuda */


/*--------------------------------------------------------------------
 * Function:    SimilarityScore
 * Purpose:     Calculate  the maximum Similarity-Score H(i,j)
 */
__global__
void similarityScore_kernel( index_t si,
                             index_t sj,
                             index_t j_upper_bound,
                             score_t* H,
                             link_t* P,
                             maxpos_t* maxPos,
                             const char* seqa,
                             const char* seqb,
                             index_t cols
                           )
{
    // compute the second loop index j
    const index_t loopj = blockIdx.x * blockDim.x + threadIdx.x;

    if (loopj >= j_upper_bound) return;

    // compute original i and j
    index_t i = si - loopj;
    index_t j = sj + loopj;

    // bounds test for matchMissmatchScore_cuda
    assert(i > 0); // was: assert(i > 0 && i <= n); -- n currently not passed in
    assert(j > 0 && j <= cols);

    // Stores index of element
    maxpos_t index = cols * i + j;

    assert(index >= cols);
    // Get element above
    score_t up = H[index - cols] + GAP_SCORE;

    assert(index > 0);
    // Get element on the left
    score_t left = H[index - 1] + GAP_SCORE;

    assert(index > cols);
    // Get element on the diagonal
    score_t diag = H[index - cols - 1] + matchMissmatchScore_cuda(i, j, seqa, seqb);

    // Calculates the maximum
    score_t max  = NONE;
    link_t  pred = NOLINK;
    /* === Matrix ===
     *      a[0] ... a[n]
     * b[0]
     * ...
     * b[n]
     *
     * generate 'a' from 'b', if '←' insert e '↑' remove
     * a=GAATTCA
     * b=GACTT-A
     *
     * generate 'b' from 'a', if '←' insert e '↑' remove
     * b=GACTT-A
     * a=GAATTCA
    */

    // same letter ↖
    if (diag > max) {
        max = diag;
        pred = DIAGONAL;
    }

    // remove letter ↑
    if (up > max) {
        max = up;
        pred = UP;
    }

    //insert letter ←
    if (left > max) {
        max = left;
        pred = LEFT;
    }

    //Inserts the value in the similarity and predecessor matrixes
    H[index] = max;
    P[index] = pred;

    // Updates maximum score to be used as seed on backtrack
    {
      // \note \pp
      //   locks seem to be a NOGO in CUDA warps,
      //   thus the update to set the maximum is made nonblocking.
      maxpos_t current = *maxPos;
      maxpos_t assumed = current+1;

      while (assumed != current && max > H[current])
      {
        assumed = current;

        // \note consider atomicCAS_system for multi GPU systems
        current = atomicCAS(maxPos, assumed, index);
      }
    }
}  /* End of similarityScore_kernel */



/*--------------------------------------------------------------------
 * Function:    main
 */
int main(int argc, char* argv[])
{
  typedef std::chrono::time_point<std::chrono::system_clock> time_point;

  bool     useBuiltInData = true;
  maxpos_t maxPos = 0;

  if (argc==3)
  {
    m = strtoll(argv[1], NULL, 10);
    n = strtoll(argv[2], NULL, 10);
    useBuiltInData = false;
  }

//#ifdef DEBUG
  if (useBuiltInData)
    printf ("Using built-in data for testing ..\n");

  // Allocates a and b
  a = (char*)malloc((m+1) * sizeof(char));
  b = (char*)malloc((n+1) * sizeof(char));
  //~ a = unified_alloc<char>(m);
  //~ b = unified_alloc<char>(n);

  std::cerr << "a,b allocated: " << m << "/" << n << std::endl;

  // Because now we have zeros
  m++;
  n++;

  if (useBuiltInData)
  {
    //Uncomment this to test the sequence available at
    //http://vlab.amrita.edu/?sub=3&brch=274&sim=1433&cnt=1
    // assert(m=11 && n=7);
    // a[0] =   'C';
    // a[1] =   'G';
    // a[2] =   'T';
    // a[3] =   'G';
    // a[4] =   'A';
    // a[5] =   'A';
    // a[6] =   'T';
    // a[7] =   'T';
    // a[8] =   'C';
    // a[9] =   'A';
    // a[10] =  'T';

    // b[0] =   'G';
    // b[1] =   'A';
    // b[2] =   'C';
    // b[3] =   'T';
    // b[4] =   'T';
    // b[5] =   'A';
    // b[6] =   'C';
    // https://en.wikipedia.org/wiki/Smith%E2%80%93Waterman_algorithm#Example
    // Using the wiki example to verify the results
    assert(m>=8 && n>=9);

    b[0] =   'G';
    b[1] =   'G';
    b[2] =   'T';
    b[3] =   'T';
    b[4] =   'G';
    b[5] =   'A';
    b[6] =   'C';
    b[7] =   'T';
    b[8] =   'A';

    a[0] =   'T';
    a[1] =   'G';
    a[2] =   'T';
    a[3] =   'T';
    a[4] =   'A';
    a[5] =   'C';
    a[6] =   'G';
    a[7] =   'G';
  }
  else
  {
    //Gen random arrays a and b
    generate();
  }
  
  

  // Allocates similarity matrix H
  score_t* H = (score_t*)malloc(m * n * sizeof(score_t));

  //Allocates predecessor matrix P
  link_t*  P = (link_t*)malloc(m * n * sizeof(link_t));

  time_point     starttime = std::chrono::system_clock::now();
  
  
  // \todo \pp
  //   gpuA and gpuB could be allocated in constant memory (if small enough)
  char*       gpuA       = shared_alloc_transfer(a, m);
  char*       gpuB       = shared_alloc_transfer(b, n);
  score_t*    gpuH       = shared_alloc_zero<score_t>(m*n);
  link_t*     gpuP       = shared_alloc_zero<link_t>(m*n);
  maxpos_t*   gpuMaxPos  = shared_alloc_zero<maxpos_t>();

  // Because now we have zeros ((m-1) + (n-1) - 1)
  long long int nDiag = m + n - 3;

  for (int i = 1; i <= nDiag; ++i)
  {
      long long nEle = nElement(i);
      long long si /* uninitialized */;
      long long sj /* uninitialized */;

      calcFirstDiagElement(i, &si, &sj);

      {
        // CUDA, here we go

        // \note
        //   * MAKE SURE THAT a,b,H,P,maxPos are ACCESSIBLE from GPU.
        //     This prototype allocates a,b,H,P in unified memory space, thus
        //     we just copy the pointers. If the allocation policy changes,
        //     memory referenced by a,b,H,P has to be transferred to the GPU,
        //     and memory referenced by H and P has to be transferred back.
        //   * a and b do not change, thus they only need to be transferred
        //     initially.
        //   * transfers of H and P could probably be optimized to only
        //     include data along the wavefront.
        // \todo
        //   * study amount of data transfer for H and P
        //~ const long long ITER_SPACE = ceil(nEle/THREADS_PER_BLOCK);
        const long long ITER_SPACE = (nEle+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;

        // comp. of ai and aj moved into CUDA kernel
        similarityScore_kernel
            <<<ITER_SPACE, THREADS_PER_BLOCK, 0, 0>>>
            (si, sj, nEle, gpuH, gpuP, gpuMaxPos, gpuA, gpuB, m);
      }
  }
  
  // data transfer
  //   P,H,maxPos
  shared_to_host_free(P,       gpuP, m*n);
  if (DEBUG_MODE || useBuiltInData)
    shared_to_host_free(H,       gpuH, m*n);
  else
    shared_free(gpuH);

  shared_to_host_free(&maxPos, gpuMaxPos);
  shared_free(gpuA);
  shared_free(gpuB);
  
  check_cuda_success( hipStreamSynchronize(0) );

  time_point     endtime = std::chrono::system_clock::now();
  int len = backtrack(P, maxPos);

  if (DEBUG_MODE)
  {
    printf("\nSimilarity Matrix:\n");
    printMatrix(H);
  
    printf("\nPredecessor Matrix:\n");
    printPredecessorMatrix(P);
  }

  if (useBuiltInData)
  { 
    const bool correct = H[maxPos] == 13;
       
    std::cerr << "Max(builtin data): " << H[maxPos] << " == 13? " << correct
              << " " << maxPos 
              << std::endl;
    
    if (!correct) throw std::logic_error("Invalid result"); 
  }

  int elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(endtime-starttime).count();

  std::cout << "\nElapsed time: " << elapsed << " ms"
            << "\nPath length: " << len 
            << "\nScore: " << (DEBUG_MODE || useBuiltInData ? H[maxPos] : -1) 
            << std::endl;

  // Frees similarity matrixes
  free(H);
  free(P);

  //Frees input arrays
  free(a);
  free(b);

  return 0;
}  /* End of main */

/*--------------------------------------------------------------------
 * Function:    nElement
 * Purpose:     Calculate the number of i-diagonal elements
 */
long long int nElement(long long int i) {
    if (i < m && i < n) {
        // Number of elements in the diagonal is increasing
        return i;
    }
    else if (i < max(m, n)) {
        //Number of elements in the diagonal is stable
        long int min_mn = min(m, n);
        return min_mn - 1;
    }
    else {
        //Number of elements in the diagonal is decreasing
        long int min_mn = min(m, n);
        return 2 * min_mn - i + abs(m - n) - 2;
    }
}

/*--------------------------------------------------------------------
 * Function:    calcElement
 * Purpose:     Calculate the position of (si, sj)-element
 */
void calcFirstDiagElement(long long int i, long long int *si, long long int *sj) {
    // Calculate the first element of diagonal
    if (i < n) {
        *si = i;
        *sj = 1;
    } else {
        *si = n - 1;
        *sj = i - n + 2;
    }
}



/*--------------------------------------------------------------------
 * Function:    backtrack
 * Purpose:     Modify matrix to print, path change from value to PATH
 */
int backtrack(link_t* P, maxpos_t maxPos) {
    //hold maxPos value
    long long int predPos = 0;
    int        len = 0;

#ifdef DEBUG
    std::cerr << "maxpos = " << maxPos << std::endl;
#endif

    //backtrack from maxPos to startPos = 0
    do {
#ifdef DEBUG
        std::cerr << "P[" << maxPos << "] = "
                  << std::flush
                  << P[maxPos]
                  << std::endl;
#endif

        switch (P[maxPos])
        {
          case DIAGONAL:
            predPos = maxPos - m - 1;
            break;

          case UP:
            predPos = maxPos - m;
            break;

          case LEFT:
            predPos = maxPos - 1;
            break;

          default:
            assert(false);
        }

#ifdef DEBUG
        P[maxPos] *= PATH;
#endif
        maxPos = predPos;
        ++len;
    } while (P[maxPos] != NONE);

    return len;
}  /* End of backtrack */

/*--------------------------------------------------------------------
 * Function:    printMatrix
 * Purpose:     Print Matrix
 */
void printMatrix(int* matrix) {
    long long int i, j;
    printf("-\t-\t");
    for (j = 0; j < m-1; j++) {
      printf("%c\t", a[j]);
    }
    printf("\n-\t");
    for (i = 0; i < n; i++) { //Lines
        for (j = 0; j < m; j++) {
          if (j==0 && i>0) printf("%c\t", b[i-1]);
          
          if ((m * i + j) == 69) printf("*");
          printf("%d\t", matrix[m * i + j]);
        }
        printf("\n");
    }

}  /* End of printMatrix */

/*--------------------------------------------------------------------
 * Function:    printPredecessorMatrix
 * Purpose:     Print predecessor matrix
 */
void printPredecessorMatrix(link_t* matrix) {
    long long int i, j, index;
    printf("    ");
    for (j = 0; j < m-1; j++) {
      printf("%c ", a[j]);
    }
    printf("\n  ");
    for (i = 0; i < n; i++) { //Lines
        for (j = 0; j < m; j++) {
          if (j==0 && i>0) printf("%c ", b[i-1]);
            index = m * i + j;
            if (matrix[index] < 0) {
                printf(BOLDRED);
                if (matrix[index] == -UP)
                    printf("↑ ");
                else if (matrix[index] == -LEFT)
                    printf("← ");
                else if (matrix[index] == -DIAGONAL)
                    printf("↖ ");
                else
                    printf("- ");
                printf(RESET);
            } else {
                if (matrix[index] == UP)
                    printf("↑ ");
                else if (matrix[index] == LEFT)
                    printf("← ");
                else if (matrix[index] == DIAGONAL)
                    printf("↖ ");
                else
                    printf("- ");
            }
        }
        printf("\n");
    }

}  /* End of printPredecessorMatrix */

/*--------------------------------------------------------------------
 * Function:    generate
 * Purpose:     Generate arrays a and b
 */
void generate() {
    //Random seed
    srand(time(NULL));

    //Generates the values of a
    long long int i;
    for (i = 0; i < m; i++) {
        int aux = rand() % 4;
        if (aux == 0)
            a[i] = 'A';
        else if (aux == 2)
            a[i] = 'C';
        else if (aux == 3)
            a[i] = 'G';
        else
            a[i] = 'T';
    }

    //Generates the values of b
    for (i = 0; i < n; i++) {
        int aux = rand() % 4;
        if (aux == 0)
            b[i] = 'A';
        else if (aux == 2)
            b[i] = 'C';
        else if (aux == 3)
            b[i] = 'G';
        else
            b[i] = 'T';
    }
} /* End of generate */


/*--------------------------------------------------------------------
 * External References:
 * http://vlab.amrita.edu/?sub=3&brch=274&sim=1433&cnt=1
 * http://pt.slideshare.net/avrilcoghlan/the-smith-waterman-algorithm
 * http://baba.sourceforge.net/
 */
